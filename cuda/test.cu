#include "cudahelpers/cuda_helpers.cuh"
#include "alg_515.cuh"

#define DEBUG

template <typename index_t>
GLOBALQUALIFIER void test_kernel(index_t N, const index_t P, index_t nCp, index_t * C)
{
  const index_t thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid >= nCp) return;
  alg515(N, P, thid, C+(thid*P));
}

int main()
{
  // integer type to use
  typedef uint32_t index_t;

  // number of elements
  constexpr index_t N = 10;
  // number of elements in each combination
  constexpr index_t P = 3;
  // binomial coefficient
  const     index_t nCp = binom(N, P);

  printf("N=%u\tP=%u\tnCp=%u\n", N, P, nCp);

  // allocate memory for result
  index_t * C_h = (index_t*)malloc(sizeof(index_t)*nCp*P);
  index_t * C_d; hipMalloc(&C_d, sizeof(index_t)*nCp*P); CUERR

  TIMERSTART(test)
  // start computation
  test_kernel<<<SDIV(nCp, 1024), 1024>>>(N, P, nCp, C_d); CUERR
  TIMERSTOP(test)

  hipMemcpy(C_h, C_d, sizeof(index_t)*nCp*P, D2H); CUERR

  //check results
  #ifdef DEBUG
  for(index_t i = 0; i < nCp; i++)
  {
    printf("[ ");
    for(index_t j = 0; j < P; j++)
    {
      printf("%u ", C_h[i*P+j]);
    }
    printf("]\n");
  }
  #endif

  free(C_h);
  hipFree(C_d); CUERR
}
