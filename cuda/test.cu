#include "cudahelpers/cuda_helpers.cuh"
#include "alg_515.cuh"

template <typename index_t>
GLOBALQUALIFIER void test_kernel(index_t N, index_t P, index_t nCp, index_t * C)
{
  const index_t thid = blockIdx.x * blockDim.x + threadIdx.x;
  if(thid >= nCp) return;
  comb(N, P, thid, C+(thid*P));
}

int main()
{
  typedef int index_t;

  constexpr index_t N = 5;
  constexpr index_t P = 3;
  const     index_t nCp = binom(N, P);

  printf("N=%u\tP=%u\tnCp=%u\n", N, P, nCp);

  index_t * C_h = (index_t*)malloc(sizeof(index_t)*nCp*P);
  index_t * C_d; hipMalloc(&C_d, sizeof(index_t)*nCp*P); CUERR

  TIMERSTART(test)
  test_kernel<<<SDIV(nCp, 1024), 1024>>>(N, P, nCp, C_d); CUERR
  TIMERSTOP(test)
  hipMemcpy(C_h, C_d, sizeof(index_t)*nCp*P, D2H); CUERR

  //check results
  for(index_t i = 0; i < nCp; i++)
  {
    printf("[ ");
    for(index_t j = 0; j < P; j++)
    {
      printf("%u ", C_h[i*P+j]);
    }
    printf("]\n");
  }


  free(C_h);
  hipFree(C_d); CUERR
}
